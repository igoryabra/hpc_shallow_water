#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <chrono>

#include "kernels.cuh" //parallel kernels
#include "utils.h"     //sequential functions

typedef std::chrono::high_resolution_clock clk;
typedef std::chrono::duration<double> second;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) getchar();
    }
}

int main(int argc, char *argv[])
{
    int Size = 500;      // Size of map, Size*Size [km]
    int nx = 2001;       // Number of cells in each direction on the grid
    double Tend = 0.20;  // Simulation time in hours [hr]
    int Nthreads = 256;  // Number of threads per block
    int nt = 0;
    double T = 0.0;

    if (argc == 3) {
        nx = atoi(argv[1]);
        Nthreads = atoi(argv[2]);
    }

    cout << "nx = " << nx << endl;
    cout << "Nthreads = " << Nthreads << endl;

    auto filename = to_string(Tend);
    filename = filename.substr(0, filename.find(".") + 2);
    filename = "../data/Data_nx" + to_string(nx) + "_" + to_string(Size) + "km_T" + filename;

    double dx = ((double)Size)/((double)nx);           // Grid spacening
    int numElements = nx * nx;                     // Total number of elements
    size_t memsize = numElements * sizeof(double); // Memory size of one array

    // Host data
    double GPU_dt, C = 0.0;
    double *H = (double*)malloc(memsize);
    double *HU = (double*)malloc(memsize);
    double *HV = (double*)malloc(memsize);
    double *Ht = (double*)malloc(memsize);
    double *HUt = (double*)malloc(memsize);
    double *HVt = (double*)malloc(memsize);
    double *Zdx = (double*)malloc(memsize);
    double *Zdy = (double*)malloc(memsize);
    double *dt = (double *)malloc(sizeof(double));
    double *GPU_mu = (double *)malloc(sizeof(int));
    read_data(filename, H, HU, HV, Zdx, Zdy, nx);;

    // Device data
    double *d_H, *d_HU, *d_HV, *d_Ht, *d_HUt, *d_HVt;
    double *d_Zdx, *d_Zdy;
    double *d_mu;
    int *d_mutex;

    gpuErrchk(hipMalloc((void **) &d_H, memsize));
    gpuErrchk(hipMalloc((void **) &d_HU, memsize));
    gpuErrchk(hipMalloc((void **) &d_HV, memsize));
    gpuErrchk(hipMalloc((void **) &d_Ht, memsize));
    gpuErrchk(hipMalloc((void **) &d_HUt, memsize));
    gpuErrchk(hipMalloc((void **) &d_HVt, memsize));
    gpuErrchk(hipMalloc((void **) &d_Zdx, memsize));
    gpuErrchk(hipMalloc((void **) &d_Zdy, memsize));
    gpuErrchk(hipMalloc((void **) &d_mu, sizeof(double)));
    gpuErrchk(hipMalloc((void **) &d_mutex, sizeof(int)));
    gpuErrchk(hipMemset(d_mu, 0.0, sizeof(double)));
    gpuErrchk(hipMemset(d_mutex, 0, sizeof(float)));

    // Copy data from host to device
    gpuErrchk(hipMemcpy(d_H, H, memsize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_HU, HU, memsize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_HV, HV, memsize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Zdx, Zdx, memsize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Zdy, Zdy, memsize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Ht,  Ht, memsize,  hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_HUt, HUt, memsize,  hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_HVt, HVt, memsize,  hipMemcpyHostToDevice));

    cpy_to(Ht,H,numElements);
    cpy_to(HUt,HU,numElements);
    cpy_to(HVt,HV,numElements);
    copy_host2device(d_H, d_HU, d_HV, H, HU, HV, memsize);
    copy_host2device(d_Ht, d_HUt, d_HVt, Ht, HUt, HVt, memsize);

    // Set grid and block dimensions
    // For finite volume kernel
    int NblocksFV = ((nx-2)*(nx-2) + Nthreads -1) / Nthreads;
    // For tolerances kernel
    int NblocksBC = pow(2, ceil(log2(nx))) / Nthreads;
    dim3 GridDimBC(NblocksBC,4);
    // For Enforce BC kernel
    int NblocksTol = (nx*nx + Nthreads -1) / Nthreads;

    const clock_t begin_time = clock();

    while (T < Tend) {
        find_mumax_kernel<<<Nthreads, Nthreads>>>(d_H, d_HU, d_HV, d_mutex, d_mu, numElements);
        hipMemcpy(GPU_mu, d_mu, sizeof(double), hipMemcpyDeviceToHost);
        GPU_dt = dx / (sqrt(2.0) * GPU_mu[0]);
        if(T + GPU_dt > Tend) {
            GPU_dt = Tend - T;
        }

        // Print status
        cout << "Computing T: " << T + GPU_dt << ". " << 100 * (T + GPU_dt) / Tend << "%\n";

        swap(d_H, d_HU, d_HV, d_Ht, d_HUt, d_HVt);
        hipDeviceSynchronize();

        enforce_BC_kernel<<<GridDimBC,Nthreads>>>(d_Ht, d_HUt, d_HVt, nx);
        hipDeviceSynchronize();

        C = (.5 * GPU_dt / dx);
        FV_iterator_kernel<<<NblocksFV,Nthreads>>>(d_H, d_HU, d_HV, d_Zdx, d_Zdy, d_Ht, d_HUt, d_HVt, C, GPU_dt, nx);
        hipDeviceSynchronize();

        impose_tolerances_kernel<<<NblocksTol,Nthreads>>>(d_H, d_HU, d_HV, numElements);
        T = T + GPU_dt;
        nt++;
        hipDeviceSynchronize();
    }

    hipMemcpy(H, d_H, memsize, hipMemcpyDeviceToHost);

    filename = to_string(Tend);
    filename = filename.substr(0, filename.find(".") + 2);
    filename = "Solution_nx" + to_string(nx) + "_" + to_string(Size) + "km_T" + filename + "_h.bin";
    write_file(filename, H, nx);

    hipFree(d_H);
    hipFree(d_HU);
    hipFree(d_HV);
    hipFree(d_Zdx);
    hipFree(d_Zdy);
    hipFree(d_Ht);
    hipFree(d_HUt);
    hipFree(d_HVt);
    hipFree(d_mu);
    hipFree(d_mutex);

    free(H);
    free(HU);
    free(HV);
    free(Zdx);
    free(Zdy);
    free(Ht);
    free(HUt);
    free(HVt);
    free(GPU_mu);

    double time = (double)(clock() - begin_time) / CLOCKS_PER_SEC;

    // Communicate time-to-compute
    int ops = nt * (15 + 2 + 11 + 30 + 30 + 1) * nx^2;
    double flops = (double)ops / time;
    cout << "Time to compute solution : " << time << " seconds\n";
    cout << "Average performance      : " << flops / 1.0e9 << " gflops\n";

    return 0;
}